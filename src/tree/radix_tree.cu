#include "hip/hip_runtime.h"
#include <cstdlib>
#include <ctime>
#include <algorithm>
#include <iostream>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include "radix_tree_generic.h"

#define N_THREADS_PER_BLK 256


class RadixTree : public RadixTreeGeneric<int, int> {
protected:
    __host__ __device__ code_t getCode(const int &x) const override {
        return x;
    }
    __host__ __device__ int startValue() const override {
        return 0;
    }
    __host__ __device__ int elementToValue(const int &x) const override {
        return x;
    }
    __host__ __device__ void update(int &dst, const int &src) const override {
        dst ^= src;
    }
    __device__ void atomicUpdate(int &dst, const int &src) const override {
        atomicXor(&dst, src);
    }
};


__global__  void init(RadixTree *tree, int n) {
    tree->init(n);
}
__global__ void construct(RadixTree *tree, int *arr) {
    tree->construct(arr);
}
__global__ void destroy(RadixTree *tree) {
    tree->destroy();
}
__global__ void check(RadixTree *tree, bool *res) {
    *res = tree->check();
}


const int N = 10, MAX = 100;
int arr[N];

int main() {
    srand(time(0));
    for (int i = 0; i < N; ++i) arr[i] = rand() % MAX;
    std::sort(arr, arr + N);
    int n = 0;
    for (int i = 0; i < N; ++i)
        if (i == 0 || arr[i] != arr[i - 1])
            arr[n++] = arr[i];
    std::random_shuffle(arr, arr + n);
    std::cout << n << std::endl;

    bool res = false;

    // CPU version
    /*
    std::sort(arr, arr + n);
    RadixTree tree;
    tree.init(n);
    tree.construct(arr);
    res = tree.check();
    tree.print();
    tree.destroy();
    */

    // GPU version
    int *arr_dev;
    bool *res_dev;
    RadixTree *tree_dev;
    hipMalloc(&arr_dev, n * sizeof(int));
    hipMalloc(&res_dev, sizeof(bool));
    hipMalloc(&tree_dev, sizeof(RadixTree));
    hipMemcpy(arr_dev, arr, n * sizeof(int), hipMemcpyHostToDevice);

    /*
    init<<<1, 1>>>(tree_dev, n);
    thrust::sort(thrust::device_ptr<int>(arr_dev), thrust::device_ptr<int>(arr_dev) + n);
    int nblks = min(64, (n + N_THREADS_PER_BLK - 1) / N_THREADS_PER_BLK);
    construct<<<nblks, N_THREADS_PER_BLK>>>(tree_dev, arr_dev);
    check<<<1, 1>>>(tree_dev, res_dev);
    hipMemcpy(&res, res_dev, sizeof(bool), hipMemcpyDeviceToHost);
    destroy<<<1, 1>>>(tree_dev);
    */

    RadixTreeWrapper<RadixTree> tw(n);
    tw.construct(arr_dev);
    tw.print();
    res = tw.check();

    hipDeviceSynchronize();
    cudaCheckError();
    hipFree(arr_dev);
    hipFree(res_dev);
    hipFree(tree_dev);

    std::cout << (res ? "Success" : "Failed") << std::endl;
    return 0;
}
