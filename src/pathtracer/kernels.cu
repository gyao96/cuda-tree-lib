#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "kernels.h"
namespace CGL
{

    __device__ int addem(int a, int b)
    {
        return a + b;
    }

    __global__ void add(int a, int b, int* c)
    {
        *c = addem(a, b);
    }


    void pathtraceInit()
    {

        int c;
        int* dev_c;
        hipMalloc((void**)&dev_c, sizeof(int));

        add << <1, 1 >> > (2, 7, dev_c);

        hipMemcpy(&c, dev_c, sizeof(int),
            hipMemcpyDeviceToHost);
        printf("2 + 7 = %d\n", c);
        hipFree(dev_c);

    }
}
